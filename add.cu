#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "freshman.h"

__global__ void sumMatrix(float * MatA,float * MatB,float * MatC,int nx,int ny)
{
    int ix=threadIdx.x+blockDim.x*blockIdx.x;
    int iy=threadIdx.y+blockDim.y*blockIdx.y;
    int idx=ix+iy*ny;
    if (ix<nx && iy<ny)
    {
      MatC[idx]=MatA[idx]+MatB[idx];
    }
}

int main(int argc,char** argv)
{
  //printf("strating...\n");
  //initDevice(0);
  int nx=1<<13;
  int ny=1<<13;
  int nxy=nx*ny;
  int nBytes=nxy*sizeof(float);

  //Malloc
  float* A_host=(float*)malloc(nBytes);
  float* B_host=(float*)malloc(nBytes);
  float* C_host=(float*)malloc(nBytes);
  float* C_from_gpu=(float*)malloc(nBytes);
  initialData(A_host,nxy);
  initialData(B_host,nxy);

  //hipMalloc
  float *A_dev=NULL;
  float *B_dev=NULL;
  float *C_dev=NULL;
  CHECK(hipMalloc((void**)&A_dev,nBytes));
  CHECK(hipMalloc((void**)&B_dev,nBytes));
  CHECK(hipMalloc((void**)&C_dev,nBytes));


  CHECK(hipMemcpy(A_dev,A_host,nBytes,hipMemcpyHostToDevice));
  CHECK(hipMemcpy(B_dev,B_host,nBytes,hipMemcpyHostToDevice));

  int dimx=argc>2?atoi(argv[1]):32;
  int dimy=argc>2?atoi(argv[2]):32;

  double iStart,iElaps;

  // 2d block and 2d grid
  dim3 block(dimx,dimy);
  dim3 grid((nx-1)/block.x+1,(ny-1)/block.y+1);
  iStart=cpuSecond();
  sumMatrix<<<grid,block>>>(A_dev,B_dev,C_dev,nx,ny);
  CHECK(hipDeviceSynchronize());
  iElaps=cpuSecond()-iStart;
  printf("GPU Execution configuration<<<(%d,%d),(%d,%d)|%f sec\n",
        grid.x,grid.y,block.x,block.y,iElaps);
  CHECK(hipMemcpy(C_from_gpu,C_dev,nBytes,hipMemcpyDeviceToHost));

  hipFree(A_dev);
  hipFree(B_dev);
  hipFree(C_dev);
  free(A_host);
  free(B_host);
  free(C_host);
  free(C_from_gpu);
  hipDeviceReset();
  return 0;
}

