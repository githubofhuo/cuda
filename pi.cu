
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void reducePI(float *d_sum, int num)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x; //线程数
    int gid = id;
    float temp;
    extern float __shared__ s_pi[]; // 动态分配长度为block的线程数
    s_pi[threadIdx.x] = 0.0f;

    while (gid < num) {
        temp = (gid + 0.5f) / num; // 当前x的值
        s_pi[threadIdx.x] += 4.0f;
        gid += blockDim.x * gridDim.x;
    }

    for (int i = (blockDim.x>>1); i > 0; i>>=1) {
        if (threadIdx.x < i) {
            s_pi[threadIdx.x] += s_pi[threadIdx.x+i];
        }
        __syncthreads();
    }
    if (threadIdx.x == 0) d_sum[blockIdx.x] = s_pi[0];
}

__global__ void reducePI2(float *d_sum,int num,float *d_pi)
{
    int id=threadIdx.x;
    extern float __shared__ s_sum[];
    s_sum[id]=d_sum[id];
    __syncthreads();
    
    for(int i = (blockDim.x>>1); i>0; i>>=1){
        if(id<i) s_sum[id]+=s_sum[id+i];
        __syncthreads();
    }
    // printf("%d,%f\n",id,s_sum[id]);
    if(id==0)
    {
    *d_pi=s_sum[0]/num;
    // printf("%d,%f\n",id,*pi);
    } 
}
int main()
{
    return 0;
}